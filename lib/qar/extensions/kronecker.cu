#include <hip/hip_runtime.h>

#include <stdio.h>
#include <math.h>

__global__ void kronecker(int *a, int *b, int *c, size_t m1, size_t n1, size_t m2, size_t n2)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t j = blockIdx.y * blockDim.y + threadIdx.y;
    if ((i < m1 * m2) && (j < n1 * n2))
    {
        size_t m = max(m1, m2), n = max(n1, n2);
        c[i * n1 * n2 + j] = a[i / m * n1 + j / n] * b[i % m * n2 + j % n];
    }
}

void printm(int *a, size_t m, size_t n)
{
    for (size_t i = 0; i < m; i++)
    {
        for (size_t j = 0; j < n; j++)
        {
            printf("%d\t", a[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}
